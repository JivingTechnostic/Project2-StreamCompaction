#include "hip/hip_runtime.h"
#include "compaction.cuh"
#include <iostream>

int maxThreadsPerBlock = 128;
hipEvent_t beginEvent;
hipEvent_t endEvent;

// global calls
void initCuda (int N) {
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
}

__global__ void naive_scan (float* in_arr, float* scan_arr, int size, int depth) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	int val = 0;

	int in_index = index;

	if (depth == 1) {
		in_index--;
	}

	if (in_index >= 0 && index < size) {
		int exp_2 = 1;
		for (int i = 1; i < depth; i++) {
			exp_2 *= 2;
		}
		val = in_arr[in_index];
		if (in_index >= exp_2) {
			val += in_arr[in_index - exp_2];
		}
	}

	if (index < size) {
		scan_arr[index] = val;
	}
}

__global__ void shared_scan (float* in_arr, float* scan_arr, int size, int depth) {
	__shared__ float in_arr_s1 [1];	//contains the lower numbers
	//__shared__ float in_arr_s2 [blockDim.x];	//contains the higher numbers

	int index = threadIdx.x + blockIdx.x * blockDim.x;

	int exp_2 = 1;
	for (int i = 1; i < depth; i++) {
		exp_2 *= 2;
	}

	float sValue = 0;

	if (index < size) {
		in_arr_s1[index] = in_arr[index];
	}
	__syncthreads();

	int in_index = index;
	if (depth == 1) {
		in_index--;
	}

	if (in_index >= 0 && index < size) {
		sValue += in_arr_s1[in_index];
		if (in_index >= exp_2) {
			sValue += in_arr_s1[in_index - exp_2];
		}
	}
	//in_arr_s2[index] = in_arr[index];
	if (index < size) {
		scan_arr[index] = sValue;
	}
	__syncthreads();
}

void cudaScan (float* in_arr, float* out_arr, int size) {
	int numBlocks = ceil(size/(float)maxThreadsPerBlock);
	int threadsPerBlock = min(size, maxThreadsPerBlock);
	
	float* arr1, * arr2;
	hipMalloc((void**)&arr1, size*sizeof(float));
	hipMalloc((void**)&arr2, size*sizeof(float));
	
	float time;
	int max_depth = ceil(log2((float)size));
	hipMemcpy(arr1, in_arr, size*sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(beginEvent, 0);
	for (int i = 1; i <= max_depth; i++) {	// not sure why it's ceil(log2(size)) but it works.
		shared_scan<<<numBlocks, maxThreadsPerBlock>>>(arr1, arr2, size, i);
		//hipDeviceSynchronize();	// taking these out causes it to fail occasionally.
		float* temp = arr1;
		arr1 = arr2;
		arr2 = temp;
	}
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);

	hipEventElapsedTime(&time, beginEvent, endEvent);
	std::cout << "cudaGPUTime for size " << size << " was " << time << "ms" << std::endl;
	

	hipMemcpy(out_arr, arr1, size*sizeof(float), hipMemcpyDeviceToHost);
}

__global__ void scatter (float* in_arr, float* temp_arr, float* scan_arr, float* out_arr, int size) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < size && temp_arr[index] == 1) {
		out_arr[(int)scan_arr[index]] = in_arr[index];
	}
}

__global__ void compute (float* in_arr, float* out_arr, int size) {
	//compute this array based on some function
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	out_arr[index] = index % 2;
}

void cudaStreamCompaction (float* in_arr, float* out_arr, int size) {
	int numBlocks = ceil(size/(float)maxThreadsPerBlock);
	int threadsPerBlock = min(size, maxThreadsPerBlock);
	float* temp_arr, *scan_arr;
	float* arr, *compact_arr;

	hipMalloc((void**)&temp_arr, size*sizeof(int));
	hipMalloc((void**)&scan_arr, size*sizeof(int));
	hipMalloc((void**)&arr, size*sizeof(float));
	hipMalloc((void**)&compact_arr, size*sizeof(float));

	hipMemcpy(arr, in_arr, size*sizeof(float), hipMemcpyHostToDevice);

	compute<<<numBlocks, threadsPerBlock>>>(arr, temp_arr, size);
	cudaScan(arr, scan_arr, size);
	scatter<<<numBlocks, threadsPerBlock>>>(arr, temp_arr, scan_arr, out_arr, size);
	
	hipMemcpy(out_arr, compact_arr, size*sizeof(float), hipMemcpyDeviceToHost);
}